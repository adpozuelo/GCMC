#include "hip/hip_runtime.h"
/* Author: adpozuelo@gmail.com
 * Version: 1.3
 * Date: 07/2021
 */

#include <stdio.h>
#include <stdlib.h>

#include "gpu.h"

extern "C" {
#include "conf.h"
#include "energy.h"
#include "mkl_vsl.h"
}

__device__ precision __distance2__(precision *r, const precision *side) {
  for (int i = 0; i < NDIM; ++i) {
    if (r[i] > 0.5) r[i] -= 1;
    if (r[i] < -0.5) r[i] += 1;
  }

  precision rd2 = 0.0;
  for (int i = 0; i < NDIM; ++i) {
    r[i] *= side[i];
    r[i] *= r[i];
    rd2 += r[i];
  }

  return rd2;
}

__device__ precision __lennard_jones__(const precision r2,
                                       const unsigned short nit,
                                       const precision *al,
                                       const precision *bl2) {
  precision r6 = (bl2[nit] / r2) * (bl2[nit] / r2) * (bl2[nit] / r2);
  return 4 * al[nit] * r6 * (r6 - 1.0);
}

__global__ void __binary_reduction__(unsigned int *natoms_nsp,
                                     precision *g_idata, precision *g_odata) {
  __shared__ precision sdata[NTHREAD];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < natoms_nsp[0])
    sdata[tid] = g_idata[i];
  else
    sdata[tid] = 0.0;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) sdata[tid] += sdata[tid + s];
    __syncthreads();
  }

  if (tid == 0) atomicAdd(g_odata, sdata[0]);
}

__global__ void energy_gpu(precision *eng, unsigned int *natoms_nsp,
                           unsigned short *itp, unsigned short *ptype,
                           precision *r, precision *side, precision *rc2,
                           precision *al, precision *bl2, precision *esrrc) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int natoms = natoms_nsp[0];
  unsigned int nsp = natoms_nsp[1];
  precision energ = 0.0;

  if (i < natoms) {
    precision rd2;
    precision rdd[NDIM];
    unsigned short nit;

    unsigned int imol = i * NDIM;
    precision xi = r[imol];
    precision yi = r[imol + 1];
    precision zi = r[imol + 2];

    for (unsigned int j = 0; j < natoms; j++) {
      if (i != j) {
        unsigned int jmol = j * NDIM;
        nit = itp[ptype[i] * nsp + ptype[j]];

        rdd[0] = xi - r[jmol];
        rdd[1] = yi - r[jmol + 1];
        rdd[2] = zi - r[jmol + 2];

        rd2 = __distance2__(rdd, side);
        if (rd2 < rc2[nit])
          energ += __lennard_jones__(rd2, nit, al, bl2) - esrrc[nit];
      }
    }
  }
  eng[i] = energ;
}

__global__ void delta_energy_gpu(unsigned int ntest, precision *eng0,
                                 precision *eng1, precision *r_test,
                                 unsigned int *natoms_nsp, unsigned short *itp,
                                 unsigned short *ptype, precision *r,
                                 precision *side, precision *rc2, precision *al,
                                 precision *bl2, precision *esrrc) {
  unsigned int j = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int natoms = natoms_nsp[0];
  unsigned int nsp = natoms_nsp[1];

  precision energ0 = 0.0;
  precision energ1 = 0.0;

  if (j < natoms) {
    precision rdd[NDIM], rddn[NDIM];
    precision rd2;
    unsigned short nit;
    if (j != ntest) {
      unsigned int jmol = j * NDIM;
      unsigned int test_mol = ntest * NDIM;
      for (int k = 0; k < NDIM; ++k) {
        rdd[k] = r[jmol + k] - r[test_mol + k];
        rddn[k] = r[jmol + k] - r_test[k];
      }

      nit = itp[ptype[ntest] * nsp + ptype[j]];

      // before movement
      rd2 = __distance2__(rdd, side);
      if (rd2 < rc2[nit])
        energ0 = __lennard_jones__(rd2, nit, al, bl2) - esrrc[nit];

      // after movement
      rd2 = __distance2__(rddn, side);
      if (rd2 < rc2[nit])
        energ1 = __lennard_jones__(rd2, nit, al, bl2) - esrrc[nit];
    }
  }
  eng0[j] = energ0;
  eng1[j] = energ1;
}

extern "C" void gpu(Configuration *cxf, const unsigned short mode) {
  CPU_TIME_INIT
  GPU_TIME_INIT

  static unsigned int *natoms_nsp_dev;
  static unsigned short *itp_dev, *ptype_dev;
  static precision *r_dev, *side_dev, *rc2_dev, *al_dev, *bl2_dev, *esrrc_dev;

  if (mode == 0) {  // Initialize GPU memory
    unsigned short nsp2 = cxf->nsp * cxf->nsp;
    unsigned short *itp_serialized =
        (unsigned short *)malloc(nsp2 * sizeof(unsigned short));
    for (int i = 0; i < cxf->nsp; ++i)
      for (int j = 0; j < cxf->nsp; ++j)
        itp_serialized[i * cxf->nsp + j] = cxf->itp[i][j];
    unsigned int natoms_nsp[2] = {cxf->natoms, cxf->nsp};
    hipSetDevice(cxf->cuda_device);
    hipMalloc((void **)&natoms_nsp_dev, 2 * sizeof(unsigned int));
    hipMalloc((void **)&itp_dev, nsp2 * sizeof(unsigned short));
    hipMalloc((void **)&ptype_dev, cxf->natoms * sizeof(unsigned short));
    hipMalloc((void **)&r_dev, cxf->natoms * NDIM * sizeof(precision));
    hipMalloc((void **)&side_dev, NDIM * sizeof(precision));
    hipMalloc((void **)&rc2_dev, cxf->nitmax * sizeof(precision));
    hipMalloc((void **)&al_dev, cxf->nitmax * sizeof(precision));
    hipMalloc((void **)&bl2_dev, cxf->nitmax * sizeof(precision));
    hipMalloc((void **)&esrrc_dev, cxf->nitmax * sizeof(precision));

    hipMemcpy(natoms_nsp_dev, natoms_nsp, 2 * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    hipMemcpy(itp_dev, itp_serialized, nsp2 * sizeof(unsigned short),
               hipMemcpyHostToDevice);
    hipMemcpy(ptype_dev, cxf->ptype, cxf->natoms * sizeof(unsigned short),
               hipMemcpyHostToDevice);
    hipMemcpy(r_dev, cxf->r, cxf->natoms * NDIM * sizeof(precision),
               hipMemcpyHostToDevice);
    hipMemcpy(side_dev, cxf->side, NDIM * sizeof(precision),
               hipMemcpyHostToDevice);
    hipMemcpy(rc2_dev, cxf->rc2, cxf->nitmax * sizeof(precision),
               hipMemcpyHostToDevice);
    hipMemcpy(al_dev, cxf->al, cxf->nitmax * sizeof(precision),
               hipMemcpyHostToDevice);
    hipMemcpy(bl2_dev, cxf->bl2, cxf->nitmax * sizeof(precision),
               hipMemcpyHostToDevice);
    hipMemcpy(esrrc_dev, cxf->esrrc, cxf->nitmax * sizeof(precision),
               hipMemcpyHostToDevice);
    CPU_TIME_STOP
  } else if (mode == 1) {  // energy_gpu
    precision *e_by_thread_dev, *total_esr_dev, total_esr;
    hipMalloc((void **)&e_by_thread_dev, cxf->natoms * sizeof(precision));
    hipMalloc((void **)&total_esr_dev, sizeof(precision));

    unsigned int nblock = cxf->natoms / NTHREAD;
    if (cxf->natoms % NTHREAD != 0) ++nblock;

    hipMemset(total_esr_dev, 0, sizeof(precision));
    CPU_TIME_STOP

    GPU_TIME_START
    energy_gpu<<<nblock, NTHREAD>>>(e_by_thread_dev, natoms_nsp_dev, itp_dev,
                                    ptype_dev, r_dev, side_dev, rc2_dev, al_dev,
                                    bl2_dev, esrrc_dev);

    __binary_reduction__<<<nblock, NTHREAD>>>(natoms_nsp_dev, e_by_thread_dev,
                                              total_esr_dev);
    GPU_TIME_STOP

    CPU_TIME_START
    hipMemcpy(&total_esr, total_esr_dev, sizeof(precision),
               hipMemcpyDeviceToHost);

    cxf->esr = total_esr / 2;
    hipFree(e_by_thread_dev);
    hipFree(total_esr_dev);
    CPU_TIME_STOP
  } else if (mode == 2) {  // move_atoms_gpu
    unsigned int ntest;
    const int harvest_size = NDIM + 1;
    precision deltae, e_before, e_after;
    double *harvest = (double *)malloc(harvest_size * sizeof(double));

    precision *r_test = (precision *)malloc(NDIM * sizeof(precision));
    precision *r_test_dev;
    hipMalloc((void **)&r_test_dev, NDIM * sizeof(precision));

    precision *e_before_by_thread_dev, *e_after_by_thread_dev, *e_before_dev,
        *e_after_dev;
    hipMalloc((void **)&e_before_by_thread_dev,
               cxf->natoms * sizeof(precision));
    hipMalloc((void **)&e_after_by_thread_dev,
               cxf->natoms * sizeof(precision));
    hipMalloc((void **)&e_before_dev, sizeof(precision));
    hipMalloc((void **)&e_after_dev, sizeof(precision));

    unsigned int nblock = (cxf->natoms + (NTHREAD - 1)) / NTHREAD;

    for (int i = 0; i < cxf->natoms; ++i) {
      cxf->ntrial++;
      vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, cxf->streamRNG, harvest_size,
                   harvest, 0, 1);
      ntest = (unsigned int)cxf->natoms * harvest[NDIM];

      for (int j = 0; j < NDIM; ++j) {
        r_test[j] = cxf->r[ntest * NDIM + j] +
                    cxf->rdmax[j] * (2 * harvest[j] - 1) / cxf->side[j];
        if (r_test[j] < 0) r_test[j] += 1;
        if (r_test[j] > 1) r_test[j] -= 1;
      }

      hipMemcpy(r_test_dev, r_test, NDIM * sizeof(precision),
                 hipMemcpyHostToDevice);
      hipMemset(e_before_dev, 0, sizeof(precision));
      hipMemset(e_after_dev, 0, sizeof(precision));
      CPU_TIME_STOP

      GPU_TIME_START
      delta_energy_gpu<<<nblock, NTHREAD>>>(
          ntest, e_before_by_thread_dev, e_after_by_thread_dev, r_test_dev,
          natoms_nsp_dev, itp_dev, ptype_dev, r_dev, side_dev, rc2_dev, al_dev,
          bl2_dev, esrrc_dev);

      __binary_reduction__<<<nblock, NTHREAD>>>(
          natoms_nsp_dev, e_before_by_thread_dev, e_before_dev);

      __binary_reduction__<<<nblock, NTHREAD>>>(
          natoms_nsp_dev, e_after_by_thread_dev, e_after_dev);
      GPU_TIME_STOP

      CPU_TIME_START
      hipMemcpy(&e_before, e_before_dev, sizeof(precision),
                 hipMemcpyDeviceToHost);
      hipMemcpy(&e_after, e_after_dev, sizeof(precision),
                 hipMemcpyDeviceToHost);

      deltae = e_after - e_before;

      if (deltae < 0.0) {
        for (int k = 0; k < NDIM; ++k) cxf->r[ntest * NDIM + k] = r_test[k];

        hipMemcpy(r_dev + ntest * NDIM, r_test, NDIM * sizeof(precision),
                   hipMemcpyHostToDevice);

        cxf->esr += deltae;
        cxf->naccept++;
      } else {
        double xi[1];
        vdRngUniform(VSL_RNG_METHOD_UNIFORM_STD, cxf->streamRNG, 1, xi, 0, 1);
        if (exp(-deltae) > xi[0]) {
          for (int k = 0; k < NDIM; ++k) cxf->r[ntest * NDIM + k] = r_test[k];

          hipMemcpy(r_dev + ntest * NDIM, r_test, NDIM * sizeof(precision),
                     hipMemcpyHostToDevice);

          cxf->esr += deltae;
          cxf->naccept++;
        }
      }
    }

    hipFree(e_before_by_thread_dev);
    hipFree(e_after_by_thread_dev);
    hipFree(r_test_dev);
    hipFree(e_before_dev);
    hipFree(e_after_dev);
    free(harvest);
    free(r_test);
    CPU_TIME_STOP
  } else if (mode == 3) {  // Release GPU memory
    hipFree(natoms_nsp_dev);
    hipFree(itp_dev);
    hipFree(ptype_dev);
    hipFree(r_dev);
    hipFree(side_dev);
    hipFree(rc2_dev);
    hipFree(al_dev);
    hipFree(bl2_dev);
    hipFree(esrrc_dev);
    CPU_TIME_STOP
  } else {
    fputs("ERROR: Incorrect GPU code!\n", stderr);
    exit(1);
  }
  GPU_TIME_DESTROY
}
